#include "hip/hip_runtime.h"
#include "ProjHelperFun.cu.h"
#include "Constants.h"
#include "InitKernels.cu.h"
#include "CoreKernels.cu.h"

////////////////////////////////////////////////////////////////////////////////
////////////////////////////DEBUGGING//////////////////////
__global__ void getList(PrivGlobsCuda* globsList, 
                                 REAL* res_out,
                                 const unsigned size
                                 // REAL* mat
){
    const unsigned int gid = threadIdx.x + blockIdx.x*blockDim.x;

    PrivGlobsCuda globs = globsList[8];
    if(gid < size){
        //res_out[gid] = globs.myVarX[gid];
        //res_out[gid] = globs.myTimeline[gid];
        // res_out[gid] = globs.myResult[gid];
        //res_out[gid] = mat[gid];
        res_out[0] = (REAL) globs.myXindex;
        res_out[1] = (REAL) globs.myYindex;
        res_out[2] = (REAL) globs.myResult[idx2d(globs.myXindex,
                                            globs.myYindex,
                                            globs.myResultCols)];
        //res_out[gid] = globs.myResult[gid];
    }
    /*
    for( unsigned j = 0; j < outer; ++ j ) { //par
        res[j] = globs[j].myResult[globs[j].myXindex][globs[j].myYindex];
    }
    */
}

////////////////////////////////////////////////////////////////////////////////



//wrapper for the kernelUpdate
void updateWrapper( PrivGlobsCuda* globsList, const unsigned g,
        const unsigned numX, const unsigned numY, const unsigned outer, 
        const REAL alpha, const REAL beta, const REAL nu
){

    //8*8*8 = 512 =< 1024
    const int x = numX;
    const int y = numY;
    const int z = outer;

    const int dimx = ceil( ((float)x) / TVAL );
    const int dimy = ceil( ((float)y) / TVAL );
    const int dimz = ceil( ((float)z) / TVAL );
    dim3 block(TVAL,TVAL,TVAL), grid(dimx,dimy,dimz);

    kernelUpdate <<< grid, block>>>(globsList, g, x, y, z, alpha, beta, nu);
    hipDeviceSynchronize();
}


void rollbackWrapper(PrivGlobsCuda* globsList, const unsigned g, 
                     const unsigned outer, const unsigned numX, 
                     const unsigned numY, const unsigned numZ
){
    // create all arrays as multidim arrays for rollback()
    REAL *u, *uT, *v, *y, *yy;
    //[3.dim][1.dim][2.dim]
    //u = [numY][numX][outer]; numY rows, numX cols
    hipMalloc((void**)&u,  outer*( numY*numX*sizeof(REAL)  ));
    hipMalloc((void**)&uT, outer*( numX*numY*sizeof(REAL)  ));
    hipMalloc((void**)&v,  outer*( numX*numY*sizeof(REAL)  ));
    hipMalloc((void**)&y,  outer*( numX*numY*sizeof(REAL)  ));
    hipMalloc((void**)&yy, outer*( numX*numY*sizeof(REAL)  ));
    // hipMalloc((void**)&yy, outer*( numX*sizeof(REAL)  ));

    REAL *a, *b, *c, *aT, *bT, *cT;
    hipMalloc((void**)&a,  outer*( numY*numX*sizeof(REAL)  ));
    hipMalloc((void**)&b,  outer*( numY*numX*sizeof(REAL)  ));
    hipMalloc((void**)&c,  outer*( numY*numX*sizeof(REAL)  ));
    hipMalloc((void**)&aT, outer*( numX*numY*sizeof(REAL)  ));
    hipMalloc((void**)&bT, outer*( numX*numY*sizeof(REAL)  ));
    hipMalloc((void**)&cT, outer*( numX*numY*sizeof(REAL)  ));

    const int x = numZ;    //max(myXsize, numY), myXsize = numX
    //const int y = numZ = x;    //max(y, myYsize), myYsize = numY

    int dimx = ceil( ((float)x) / TVAL );
    int dimy = ceil( ((float)x) / TVAL );
    int dimz = outer;
    dim3 block(TVAL,TVAL,1), grid(dimx,dimy,dimz);

    const unsigned n = numY*numX;
    unsigned int block_size = 512;
    unsigned int num_blocks = (n + (block_size - 1)) / block_size;
    unsigned int sh_mem_size = block_size * 32;

    kernelRollback1 <<< grid, block >>> (   globsList, g, outer, 
                                            u, uT, v, y, 
                                            a, b, c, aT, bT, cT);
    hipDeviceSynchronize();


    transpose3dTiled<TVAL><<< grid, block >>>(uT, u, numY, numX);
    hipDeviceSynchronize();

    transpose3dTiled<TVAL><<< grid, block >>>(aT, a, numX, numY);
    hipDeviceSynchronize();
    transpose3dTiled<TVAL><<< grid, block >>>(bT, b, numX, numY);
    hipDeviceSynchronize();
    transpose3dTiled<TVAL><<< grid, block >>>(cT, c, numX, numY);  
    hipDeviceSynchronize();

    //Tridag 1
    //tridag1(outer, u, yy, a, b, c, numX, numY, numZ);
    kernelTridag1 <<< num_blocks, block_size, sh_mem_size >>> 
                                    (outer, u, yy, a, b, c, numX, numY);
    hipDeviceSynchronize();

    kernelRollback2 <<< grid, block>>> (    globsList, g, outer, 
                                            u, uT, v, y, yy, 
                                            a, b, c, aT, bT, cT);
    hipDeviceSynchronize();

    transpose3dTiled<TVAL><<< grid, block >>>(aT, a, numY, numX);
    hipDeviceSynchronize();
    transpose3dTiled<TVAL><<< grid, block >>>(bT, b, numY, numX);
    hipDeviceSynchronize();
    transpose3dTiled<TVAL><<< grid, block >>>(cT, c, numY, numX);
    hipDeviceSynchronize();
    transpose3dTiled<TVAL><<< grid, block >>>(u, uT, numX, numY);
    hipDeviceSynchronize();

    kernelRollback3 <<< grid, block>>> (globsList, g, outer, uT, v, y);
    hipDeviceSynchronize();

    //tridag2(globsList, outer, y, yy, aT, bT, cT, numX, numY, numZ);
    kernelTridag2 <<< num_blocks, block_size, sh_mem_size >>> 
                        (globsList, outer, y, yy, aT, bT, cT, numX, numY);
    hipDeviceSynchronize();

    {
        //unsigned s = numX*numY;
        unsigned size = 3;
        unsigned mem_size = size*sizeof(REAL);

        unsigned num_threads = size;
        unsigned block_size = 512;
        unsigned int num_blocks = ceil(((float) num_threads) / block_size);

        REAL *res, *d_res;
        hipMalloc((void**)&d_res, mem_size);
        res = (REAL*) malloc(mem_size);

        getList<<< num_blocks, block_size>>>(globsList, d_res, size);
        hipDeviceSynchronize();

        hipMemcpy(res, d_res, mem_size, hipMemcpyDeviceToHost);

        printf("\nres = [\n");
        for(unsigned i=0; i < size; i++)
            printf("[%d] = %.5f\n", i, res[i]);
        printf("\n]\n");

        //exit(0);
    }

    hipFree(u);
    hipFree(uT);
    hipFree(v);
    hipFree(y);
    hipFree(yy);
    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(aT);
    hipFree(bT);
    hipFree(cT);
}

void getResultsWrapper(PrivGlobsCuda* globsList, 
                       const unsigned outer, 
                       REAL* res){
    const unsigned int num_threads  = outer;
    const unsigned int block_size   = 512;
    unsigned int num_blocks         = ceil(((float) num_threads) / block_size);

    unsigned int mem_size           = outer * sizeof(REAL);

    //(*res) = (REAL*) malloc(mem_size);
    {
        float* d_out;
        hipMalloc((void**)&d_out, mem_size);
    
        kernelGetResults<<< num_blocks, block_size>>> (globsList, d_out, outer);
        hipDeviceSynchronize();
        
        //cuda results to mem
        hipMemcpy(res, d_out, mem_size, hipMemcpyDeviceToHost);
        hipFree(d_out);
    }
}






void   run_GPU(
                const unsigned int&   outer,
                const unsigned int&   numX,
                const unsigned int&   numY,
                const unsigned int&   numT,
                const REAL&           s0,
                const REAL&           t,
                const REAL&           alpha,
                const REAL&           nu,
                const REAL&           beta,
                      REAL*           res   // [outer] RESULT
) {
 
    // sequential loop distributed.
    PrivGlobsCuda* globsList;
    const unsigned numZ = max(numX, numY);
    //hipMalloc((void**)&globsList, outer*sizeof(struct PrivGlobsCuda));

    printf("init begin\n");
    init(&globsList, outer, s0, alpha, nu, t, numX, numY, numT);
    printf("init done\n");
    



    ///////////////////////////////////////////////////

// {
//         //unsigned s = numX*numY;
//         unsigned size = 4;
//         unsigned mem_size = size*sizeof(REAL);

//         unsigned num_threads = size;
//         unsigned block_size = 512;
//         unsigned int num_blocks = ceil(((float) num_threads) / block_size);

//         REAL *res, *d_res;
//         hipMalloc((void**)&d_res, mem_size);
//         res = (REAL*) malloc(mem_size);

//         getList<<< num_blocks, block_size>>>(globsList, d_res, size);
//         hipDeviceSynchronize();

//         hipMemcpy(res, d_res, mem_size, hipMemcpyDeviceToHost);

//         printf("\nres = [\n");
//         for(unsigned i=0; i < size; i++)
//             printf("[%d] = %.5f\n", i, res[i]);
//         printf("\n]\n");

//         //exit(0);
//     }

    //////////////////////////////////////////////////////

    
    
    for(int g = numT-2;g>=0;--g){ //seq
        //updateParams()
        printf("update begin\n");
        updateWrapper(globsList, g, numX, numY, outer, alpha, beta, nu);
        printf("update done\n");
        //rollback()
        printf("rollback begin\n");
        rollbackWrapper(globsList, g, outer, numX, numY, numZ);
        printf("rollback done\n");
    }
    getResultsWrapper(globsList, outer, res);
}


//#endif // PROJ_CORE_ORIG
