#include "hip/hip_runtime.h"
#include "ProjHelperFun.cu.h"

/**************************/
/**** HELPER FUNCTIONS ****/
/**************************/

/**
 * Fills in:
 *   globs.myTimeline  of size [0..numT-1]
 *   globs.myX         of size [0..numX-1]
 *   globs.myY         of size [0..numY-1]
 * and also sets
 *   globs.myXindex and globs.myYindex (both scalars)
 */
void initGrid(  const REAL s0, const REAL alpha, const REAL nu,const REAL t,
                const unsigned numX, const unsigned numY, const unsigned numT, PrivGlobs& globs
) {
    // Can be parallelized directly as each iteration writes to independent
    // globs.myTimeline indices
    for(unsigned i=0;i<numT;++i)  // par
        globs.myTimeline[i] = t*i/(numT-1);

    const REAL stdX = 20.0*alpha*s0*sqrt(t);
    const REAL dx = stdX/numX;
    globs.myXindex = static_cast<unsigned>(s0/dx) % numX;

    // Can be parallelized directly as each iteration writes to independent
    // globs.myX indices.
    for(unsigned i=0;i<numX;++i)  // par
        globs.myX[i] = i*dx - globs.myXindex*dx + s0;

    const REAL stdY = 10.0*nu*sqrt(t);
    const REAL dy = stdY/numY;
    const REAL logAlpha = log(alpha);
    globs.myYindex = static_cast<unsigned>(numY/2.0);

    // Can be parallelized directly as each iteration writes to independent
    // globs.myY indices.
    for(unsigned i=0;i<numY;++i)  // par
        globs.myY[i] = i*dy - globs.myYindex*dy + logAlpha;
}

/**
 * Fills in:
 *    Dx  [0..n-1][0..3] and
 *    Dxx [0..n-1][0..3]
 * Based on the values of x,
 * Where x's size is n.
 */
void initOperator(  const vector<REAL>& x,
                    vector<vector<REAL> >& Dxx
) {
    const unsigned n = x.size();

    REAL dxl, dxu;

    //  lower boundary
    dxl      =  0.0;
    dxu      =  x[1] - x[0];

    Dxx[0][0] =  0.0;
    Dxx[0][1] =  0.0;
    Dxx[0][2] =  0.0;
    Dxx[0][3] =  0.0;

    //  standard case
    // Can be parallelized directly as each iteration writes to independent
    // Dxx indices. x is only read, so each iteration is independent.
    // x could be put in shared memory.
    for(unsigned i=1;i<n-1;i++) // par
    {
        dxl      = x[i]   - x[i-1];
        dxu      = x[i+1] - x[i];

        Dxx[i][0] =  2.0/dxl/(dxl+dxu);
        Dxx[i][1] = -2.0*(1.0/dxl + 1.0/dxu)/(dxl+dxu);
        Dxx[i][2] =  2.0/dxu/(dxl+dxu);
        Dxx[i][3] =  0.0;
    }

    //  upper boundary
    dxl        =  x[n-1] - x[n-2];
    dxu        =  0.0;

    Dxx[n-1][0] = 0.0;
    Dxx[n-1][1] = 0.0;
    Dxx[n-1][2] = 0.0;
    Dxx[n-1][3] = 0.0;
}


void transposeVect(vector<vector<REAL> > MIn,
               vector<vector<REAL> >* MOut,
               unsigned int M,
               unsigned int N) 
{
    for (int i = 0 ; i < M ; i++) {
        for (int j = 0 ; j < N ; j++) {
            (*MOut)[i][j] = MIn[j][i];
       }
   }
}

void transpose(REAL* A, REAL** B, int M, int N) {
    for(int i = 0 ; i < M ; i++) {
        for(int j = 0 ; j < N ; j++) {
            //(*B)[j*M+i] = A[i*N+j];
            (*B)[i*N+j] = A[j*M+i];
        }
    }
}

// row = row idx
// col = col idx
// width = number of columns in the matrix
// ex: A[row,col] = A[idx2d(row, col, a.cols)]
unsigned int idx2d(int row, int col, int width) {
    return row * width + col;
}
